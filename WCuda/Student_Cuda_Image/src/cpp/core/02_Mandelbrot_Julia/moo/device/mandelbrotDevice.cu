#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
#include "MandelbrotMath.h"

using namespace gpu;

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath)
    {
    MandelbrotMath mandelbrotMath = MandelbrotMath(t);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i, j;
    double x, y;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	domaineMath.toXY(i, j, &x, &y);

	mandelbrotMath.colorXY(&ptrDevPixels[s], x, y, t);
	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

