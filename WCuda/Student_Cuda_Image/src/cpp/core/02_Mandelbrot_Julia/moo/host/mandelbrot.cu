#include "hip/hip_runtime.h"
#include "mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(uint nMin, uint nMax, const Grid& grid, uint w, uint h, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4", domaineMath), variateurAnimation(Interval<float>(nMin, nMax), 1.f)
    {
    this->t = nMin;
    }

Mandelbrot::~Mandelbrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("mandelbrot rgba uchar4 (before kernel)");

    t = variateurAnimation.get();
    mandelbrot <<<dg,db>>>(ptrDevPixels,w,h,t,domaineMath);

    // le kernel est importer ci-dessus (ligne 19)

    Device::synchronize(); // Temp,debug, only for printf in  GPU
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
    t = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

