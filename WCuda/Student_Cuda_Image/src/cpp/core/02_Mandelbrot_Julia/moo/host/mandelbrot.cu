#include "hip/hip_runtime.h"
#include "mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(uint nMin, uint nMax, const Grid& grid, uint w, uint h, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4", domaineMath), variateurAnimation(Interval<float>(nMin, nMax), 1.f)
    {
    // Tools
    this->t = nMin; // protected dans Animable
    }

Mandelbrot::~Mandelbrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("mandelbrot rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    t = variateurAnimation.get();
    mandelbrot <<<dg,db>>>(ptrDevPixels,w,h,t,domaineMath);

    // le kernel est importer ci-dessus (ligne 19)

    Device::lastCudaError("mandelbrot rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release

    Device::synchronize(); // Temp,debug, only for printf in  GPU
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
    t = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

